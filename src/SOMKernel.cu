#include "hip/hip_runtime.h"
#ifndef _SOMKERNELS_
#define _SOMKERNELS_

#include "math/Random.h"
#include "math/Functions.h"
#include "Functors.h"
#include "SOMNetGPU.h"

#include <cfloat>
#include <cassert>
#include <cmath>
#include <algorithm>

#include <omp.h>

#include <thrust/extrema.h>
#include <thrust/distance.h>
#include <thrust/device_vector.h>

using namespace ANNGPGPU;


typedef float (*pDistanceFu) (float, float);
__device__ pDistanceFu pBubble 		= ANN::fcn_bubble_nhood; 
__device__ pDistanceFu pGaussian 	= ANN::fcn_gaussian_nhood; 
__device__ pDistanceFu pCutGauss 	= ANN::fcn_cutgaussian_nhood; 
__device__ pDistanceFu pMexican 	= ANN::fcn_mexican_nhood; 
__device__ pDistanceFu pEpanech 	= ANN::fcn_epanechicov_nhood;

bool SOMNetGPU::AssignDistanceFunction() {
	pDistanceFu hBubble; 
	pDistanceFu hGaussian; 
	pDistanceFu hCutGauss; 
	pDistanceFu hMexican; 
	pDistanceFu hEpanech;

	hipMemcpyFromSymbol(&hBubble, HIP_SYMBOL(pBubble), sizeof(pDistanceFu) );
	hipMemcpyFromSymbol(&hGaussian, HIP_SYMBOL(pGaussian), sizeof(pDistanceFu) );
	hipMemcpyFromSymbol(&hCutGauss, HIP_SYMBOL(pCutGauss), sizeof(pDistanceFu) );
	hipMemcpyFromSymbol(&hMexican, HIP_SYMBOL(pMexican), sizeof(pDistanceFu) );
	hipMemcpyFromSymbol(&hEpanech, HIP_SYMBOL(pEpanech), sizeof(pDistanceFu) );

	if (strcmp (GetDistFunction()->name, "gaussian") == 0) {
		GetDistFunction()->distance = hGaussian;
	} else if (strcmp (GetDistFunction()->name, "mexican") == 0) {
		GetDistFunction()->distance = hMexican;
	} else if (strcmp (GetDistFunction()->name, "bubble") == 0) {
		GetDistFunction()->distance = hBubble;
	} else if (strcmp (GetDistFunction()->name, "cutgaussian") == 0) {
		GetDistFunction()->distance = hCutGauss;
	} else if (strcmp (GetDistFunction()->name, "epanechicov") == 0) {
		GetDistFunction()->distance = hEpanech;
	} else {
		printf("No preimplemented function recognized. No assignment done.");
		return 0;
	}
	printf("Preimplemented function recognized. Assignment done.");
	return 1;
}

bool SOMNetGPU::DeassignDistanceFunction() {
	if (strcmp (GetDistFunction()->name, "gaussian") == 0) {
		GetDistFunction()->distance = ANN::fcn_gaussian_nhood; 
	} else if (strcmp (GetDistFunction()->name, "mexican") == 0) {
		GetDistFunction()->distance = ANN::fcn_mexican_nhood; 
	} else if (strcmp (GetDistFunction()->name, "bubble") == 0) {
		GetDistFunction()->distance = ANN::fcn_bubble_nhood;
	} else if (strcmp (GetDistFunction()->name, "cutgaussian") == 0) {
		GetDistFunction()->distance = ANN::fcn_cutgaussian_nhood;
	} else if (strcmp (GetDistFunction()->name, "epanechicov") == 0) {
		GetDistFunction()->distance = ANN::fcn_epanechicov_nhood;
	} else {
		printf("No preimplemented function recognized. No deassignment done.");
		return 0;
	}
	printf("Preimplemented function recognized. Deassignment done.");
	return 1;
}

// new reference implementation
ANNGPGPU::BMUExport hostGetMin(std::vector<ANNGPGPU::BMUExport> &vec) {
	assert(vec.size() > 0);
	if(vec.size() > 1) {
		std::sort(vec.begin(), vec.end() );
	}
	return *vec.begin();
}

// fast when maps are big
std::pair<float, unsigned int> devGetMin(const thrust::device_vector<float> &vec) {
	thrust::device_vector<float>::const_iterator d_min = thrust::min_element(vec.begin(), vec.end() );
	unsigned int iID = thrust::distance(vec.begin(), d_min);
	return std::pair<float, unsigned int>(*d_min, iID);
}

//////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////
/*
 * Layout of SOMEdgeF2DArray:
 * 		COL1	COL2	COL3	COL(n+1)
 * ROW1		toNeur1	toNeur1	toNeur1	..
 * ROW2		toNeur2	toNeur2	toNeur2	..
 * ROW3		toNeur3	toNeur3	toNeur3	..
 * ROW(n+1)	..		..		..
 */
BMUExport
hostSOMFindBMNeuronID(std::vector<SplittedNetExport*> &SExp,
		const float &fConscRate)
{
	BMUExport resBMU;
	std::vector<ANNGPGPU::BMUExport> vBMUExp(SExp.size() );

	assert(SExp.size() > 0);
	assert(vBMUExp.size() == SExp.size() );

	omp_set_num_threads(SExp.size() );  							// create as many CPU threads as there are CUDA devices
	#pragma omp parallel 									// for(int iDevID = 0; iDevID < static_cast<int>(SExp.size() ); iDevID++) {
	{
		unsigned int iDevID 	= omp_get_thread_num();
		checkCudaErrors(hipSetDevice(iDevID) );
		
		unsigned int iWidth 	= SExp.at(iDevID)->f2dEdges.GetW();
		unsigned int iHeight 	= SExp.at(iDevID)->f2dEdges.GetH();

		assert(iWidth 	> 0);
		assert(iHeight 	> 0);

		thrust::device_vector<float> dvRes(iWidth, 0.f);

		for(int y = 0; y < static_cast<int>(iHeight); y++) {               
			thrust::transform(SExp.at(iDevID)->f2dEdges.GetRowBegin(y),
				SExp.at(iDevID)->f2dEdges.GetRowEnd(y),
				dvRes.begin(),
				dvRes.begin(),
				spowAmXpY_functor((*SExp.at(iDevID)->dvInput)[y]) );
		}

		if(fConscRate > 0.f) { 								// Implementation of conscience mechanism
			thrust::transform(dvRes.begin(),					// input
				dvRes.end(),							// input
				SExp.at(iDevID)->dvConscience->begin(),				// input
				dvRes.begin(),							// result
				sXmAmY_functor(1.f/(float)iWidth) );				// functor

			thrust::transform(dvRes.begin(),					// input
				dvRes.end(),							// input
				SExp.at(iDevID)->dvConscience->begin(),				// input
				SExp.at(iDevID)->dvConscience->begin(),				// result
				sAXmY_functor(fConscRate) );					// functor
		}

		std::pair<float, unsigned int> pCurBMUVal = devGetMin(dvRes);
		BMUExport BMU(pCurBMUVal.first, pCurBMUVal.second, iDevID);
		vBMUExp[iDevID] = BMU;
	}

	resBMU = hostGetMin(vBMUExp);
	checkCudaErrors(hipSetDevice(resBMU.iDeviceID) );
	resBMU.dvBMUPos = SExp.at(resBMU.iDeviceID)->f2dPositions.GetSubArrayY(resBMU.iBMUID);

	return resBMU;
}

/*
 * Layout of SOMPositionF2DArray:
 * 		COL1	COL2	COL3	COL(n+1)
 * ROW1		Xpos	Xpos	Xpos	..
 * ROW2		Ypos	Ypos	Ypos	..
 * ROW3		Zpos	Zpos	Zpos	..
 * ROW(n+1)	..		..		..		..
 */
template<typename BinaryFunction>
void hostSOMPropagateBW( std::vector<SplittedNetExport*> &SExp,
		const BMUExport &BMU,
		const float &fSigmaT,
		const float &fLearningRate,
		BinaryFunction binaryDistFunc
		)
{
	omp_set_num_threads(SExp.size() );  							// create as many CPU threads as there are CUDA devices
	#pragma omp parallel 									// for(int iDev = 0; iDev < static_cast<int>(SExp.size() ); iDev++) {
	{
		unsigned int iDevID 	= omp_get_thread_num();
		checkCudaErrors(hipSetDevice(iDevID) );
		
		unsigned int iWidth 	= SExp.at(iDevID)->f2dPositions.GetW();
		unsigned int iHeight 	= SExp.at(iDevID)->f2dPositions.GetH();

		thrust::device_vector<float> dvTmp(iWidth, pow(fSigmaT, 2) ); 			// temporary
		thrust::device_vector<float> dvInfl(iWidth, 0.f);
		thrust::device_vector<float> dvDist(iWidth, 0.f);

		// 1. Calc distances for all neurons to BMNeuron: Distance = sqrt(pow(x,2)+pow(y,2)+pow(z,2)+pow(n+1,2) )
		for(int y = 0; y < static_cast<int>(iHeight); y++) { 				// for each coordinate position of the neuron
			thrust::transform(
				SExp.at(iDevID)->f2dPositions.GetRowBegin(y),
				SExp.at(iDevID)->f2dPositions.GetRowEnd(y),
				dvDist.begin(),
				dvDist.begin(),
				spowAmXpY_functor(BMU.dvBMUPos[y]) );
		}

		// 2. Calculate the influence for each neuron
		thrust::transform( dvDist.begin(),						// input
			dvDist.end(), 								// input
			dvInfl.begin(), 							// result
			binaryDistFunc );							// functor

		// 3. Only handle neurons in radius:
		// 3a. Make stencil
		thrust::transform( dvDist.begin(), 						// input
			dvDist.end(),								// input
			dvTmp.begin(),								// input
			dvTmp.begin(), 								// result
			thrust::less<float>() 							// functor
		);
		// 3b. Use stencil to modify only neurons inside the radius
		iWidth 	= SExp.at(iDevID)->f2dEdges.GetW();
		iHeight = SExp.at(iDevID)->f2dEdges.GetH();
		for(int y = 0; y < static_cast<int>(iHeight); y++) {				// for each edge of the neuron
			thrust::transform_if( SExp.at(iDevID)->f2dEdges.GetRowBegin(y),		// input 1
				SExp.at(iDevID)->f2dEdges.GetRowEnd(y), 			// input 1
				dvInfl.begin(),							// input 2
				dvTmp.begin(),							// stencil
				SExp.at(iDevID)->f2dEdges.GetRowBegin(y), 			// result
				hebbian_functor(fLearningRate, (*SExp.at(iDevID)->dvInput)[y]), // functor
				thrust::identity<int>() ); 					// predicate
		}
	}
}

void hostSOMTrainHelper( std::vector<SplittedNetExport*> &SExp,
		const ANN::TrainingSet &InputSet,
		const unsigned int &iCycles,
		const float &fSigma0, 
		const float &fLearningRate0,
		const float &fConscRate,
		const ANN::DistFunction &DistFunc, 
		const unsigned int &iPatternID,
		const unsigned int &iCycle) 
{
	assert(iPatternID < InputSet.GetNrElements() );

	float fLambda 		= iCycles / log(fSigma0);
	float fSigmaT 		= fSigma0;
	float fLearningRate 	= fLearningRate0;

	// Set Input
	std::vector<float> vCurInput = InputSet.GetInput(iPatternID);
	for(int iDevID = 0; iDevID < static_cast<int>(SExp.size() ); iDevID++) {
		checkCudaErrors(hipSetDevice(iDevID) );

		thrust::device_vector<float> *p_dvInputVector = new thrust::device_vector<float>(vCurInput.size() );
		thrust::copy(vCurInput.begin(), vCurInput.end(), p_dvInputVector->begin() );
		SExp[iDevID]->dvInput = p_dvInputVector;
	}

	// Calc fSigmaT if conscience is _not_ used
	fSigmaT 	= DistFunc.rad_decay(fSigma0, iCycle, fLambda); 		// SM 1.3
	fLearningRate 	= DistFunc.lrate_decay(fLearningRate0, iCycle, iCycles); 	// SM 1.3

	// Find BMNeuron 
	BMUExport BMUExp = hostSOMFindBMNeuronID(SExp, fConscRate);

	// Propagate BW SM 2.0
	hostSOMPropagateBW( SExp,
		BMUExp,									// const
		fSigmaT,								// const
		fLearningRate,								// const
		sm20distance_functor(fSigmaT, DistFunc.distance)); 			// const
}

void hostSOMTraining( std::vector<SplittedNetExport*> &SExp,
		const ANN::TrainingSet &InputSet,
		const unsigned int &iCycles,
		const float &fSigma0, 
		const float &fLearningRate0,
		const float &fConscRate,
		const ANN::DistFunction &DistFunc,
		const ANN::TrainingMode &eMode )
{
	int iMin 		= 0;
	int iMax 		= InputSet.GetNrElements()-1;
	int iProgCount 		= 1;

	for(int iCycle = 0; iCycle < static_cast<int>(iCycles); iCycle++) {
		if(iCycles >= 10) {
			if(((iCycle+1) / (iCycles/10)) == iProgCount && (iCycle+1) % (iCycles/10) == 0) {
				std::cout<<"Current training progress calculated by the GPU is: "<<iProgCount*10.f<<"%/Step="<<iCycle+1<<std::endl;
				iProgCount++;
			}
		} 
		else {
			std::cout<<"Current training progress calculated by the CPU is: "<<(float)(iCycle+1.f)/(float)iCycles*100.f<<"%/Step="<<iCycle+1<<std::endl;
		}

		if(eMode == ANN::ANRandomMode) {
			unsigned int iRandID = ANN::RandInt(iMin, iMax);
			hostSOMTrainHelper(SExp, InputSet, iCycles, fSigma0, fLearningRate0, fConscRate, DistFunc, iRandID, iCycle);
		}
		// The input vectors are presented to the network in serial order
		else if(eMode == ANN::ANSerialMode) {
			for(unsigned int j = 0; j < InputSet.GetNrElements(); j++) {
				hostSOMTrainHelper(SExp, InputSet, iCycles, fSigma0, fLearningRate0, fConscRate, DistFunc, j, iCycle);
			}
		}
	}
}

#endif
